#include "hip/hip_runtime.h"
#include <stdio.h>
#include "matrix.h"

/*
struct hipComplex {
  float Re; float Im;
  // constructors
  __device__ hipComplex()                             : Re(0.), Im(0.) {}
  __device__ hipComplex( float a, float b )           : Re( a), Im( b) {}
  // functions
  __device__ float     abs2( void )                  { return Re*Re + Im*Im; }
  __device__ hipComplex operator+(const hipComplex& a) { return hipComplex( Re+a.Re, Im+a.Im  ); }
  __device__ hipComplex operator-(const hipComplex& a) { return hipComplex( Re-a.Re, Im-a.Im  ); }
  __device__ hipComplex operator*(const hipComplex& a) { return hipComplex( Re*a.Re - Im*a.Im, 
                                                                         Im*a.Re + Re*a.Im ); }
  __device__ hipComplex operator*(const float a)      { return hipComplex( Re*a,  Im*a       ); }
};
*/

__device__ hipComplex conj(hipComplex a) {
  hipComplex ca(a.Re,-a.Im); return ca;
}
__device__ hipComplex inv(hipComplex a) {
  float c2 = a.abs2();
  hipComplex c(a.Re/c2,-a.Im/c2)    ; return c;
}


__global__ void view_m(hipComplex X[Nc][Nc]) {
  hipComplex tmp;
  for (int i=0; i<Nc; i++) {
    for (int j=0; j<Nc; j++) {
      tmp = X[i][j]; 
      printf("   (%.3f, %.3f) ", tmp.Re, tmp.Im );
    } printf("\n");
  }   printf("\n");
}

__device__ void copy_m(hipComplex A[Nc][Nc],  hipComplex B[Nc][Nc]) {
  for (int i=0; i<Nc; i++) 
    for (int j=0; j<Nc; j++) 
      B[i][j] = A[i][j];
}

__device__ void mul_m(hipComplex A[Nc][Nc],  hipComplex B[Nc][Nc], 
                                    hipComplex C[Nc][Nc]) {
  hipComplex Ct[Nc][Nc];
  for (int i=0; i<Nc; i++) 
    for (int j=0; j<Nc; j++) 
      for (int k=0; k<Nc; k++) Ct[i][j] = Ct[i][j] + A[i][k]*B[k][j];
  for (int i=0; i<Nc; i++) 
    for (int j=0; j<Nc; j++) 
      C[i][j] = Ct[i][j];
}

__device__ void add_m(hipComplex A[Nc][Nc],  hipComplex B[Nc][Nc], 
                                    hipComplex C[Nc][Nc]) {
  for (int i=0; i<Nc; i++) 
    for (int j=0; j<Nc; j++) C[i][j] = A[i][j]+B[i][j];
}

__device__ void sub_m(hipComplex A[Nc][Nc],  hipComplex B[Nc][Nc], 
                                    hipComplex C[Nc][Nc]) {
  for (int i=0; i<Nc; i++) 
    for (int j=0; j<Nc; j++) C[i][j] = A[i][j]-B[i][j];
}

__device__ void equ_m( hipComplex a,
                                    hipComplex A[Nc][Nc]) {
  hipComplex z(0.,0.);
  for (int i=0; i<Nc; i++) 
    for (int j=0; j<Nc; j++) A[i][j] = ((i==j) ? a : z);
}


__device__ void dag_m(hipComplex A[Nc][Nc]) {
  hipComplex temp;
  for (int i=0; i<Nc; i++)
    for (int j=0; j<=i; j++) {
      temp = A[i][j];
      A[i][j] = conj(A[j][i]);
      A[j][i] = conj(temp);
    }
}

__device__ hipComplex det_m(hipComplex A[Nc][Nc]) {
  hipComplex d;
  hipComplex zer[Nc][Nc];
  hipComplex Ap[Nc][Nc];
  add_m( zer, A, Ap);
  for (int i=0; i<Nc-1; i++)
    for (int j=i+1; j<Nc; j++)
      // subtract ~ row i from row j so that A[j][i] vanishes ...
      for (int k=i+1;k<Nc;k++) { Ap[j][k] = Ap[j][k] - ( Ap[j][i] * Ap[i][k]
          )*inv(Ap[i][i]); }
  d = Ap[0][0];
  for (int i=1; i<Nc; i++) d= d * Ap[i][i];
  return d;
}

__device__ hipComplex trace( hipComplex A[Nc][Nc] ) {
  hipComplex tr = A[0][0];
  for (int i=1; i<Nc; i++) tr = tr + A[i][i];
  return tr;
}

__device__ void gramschmidt(hipComplex A[Nc][Nc]) {
  hipComplex dot;
  float norm;
  for (int i=0; i<Nc; i++) {
    // normalise ith row
    /*norm = cabs( A[i][0] ); norm = norm * norm;*/
    norm = A[i][0].abs2();
    /*for (int j=1; j<Nc; j++) norm      = norm + cabs( A[i][j] )*cabs( A[i][j] );*/
    for (int j=1; j<Nc; j++) norm       += A[i][j].abs2();
    norm = 1./sqrt(norm);
    for (int j=0; j<Nc; j++) A[i][j]   = A[i][j]*norm;
    // orthogonalise the rest
    for (int k=i+1; k<Nc; k++) {
      dot = conj(A[i][0])*(A[k][0]);
      for (int j=1; j<Nc; j++) dot     = dot + conj(A[i][j])*(A[k][j]);
      for (int j=0; j<Nc; j++) A[k][j] = A[k][j] - dot*A[i][j];
    }
  }
}

__global__ void suN_m(hipComplex A[Nc][Nc]) {
  gramschmidt( A );
  hipComplex d; 
  /*int j, k;*/
  switch (Nc) { 
    // can do Nc=2,3 by hand
    case 2: 
      A[1][0] = conj(A[0][1])*(-1.);
      A[1][1] = conj(A[0][0])*(+1.);
      break;
    case 3:
      A[2][0] = conj( A[0][1]*A[1][2] - A[1][1]*A[0][2] );
      A[2][1] = conj( A[0][2]*A[1][0] - A[1][2]*A[0][0] );
      A[2][2] = conj( A[0][0]*A[1][1] - A[1][0]*A[0][1] );
      break; 
    default:
      d = det_m(A);
      for (int i=0; i<Nc; i++) A[0][i] = A[0][i] * conj(d);
  }
}

