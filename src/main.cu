/*
 * Author: greg jackson
 * Date: Dec 02 2015
 * simple gauge theories on d = {2,3,4} lattice
 *
 *
 */

#include <stdio.h>
#include "core.h"

/*-----------------------------------------------------------------------------------------------*/

/* external parameters */

int        calls = 1000                     ;   // MC calls
int        zn    = 2                       ;   // if 0 --> U(1)
Group     *ulinks                          ;   // the lattice


void eval_Zn  ( double Bi, double Bf);
void eval_U1  ( double Bi, double Bf);
void eval_SUn ( double Bi, double Bf);

void iter_SUn( double beta, int cls);
void wilson(   double beta );

/*-----------------------------------------------------------------------------------------------*/

int main() {

  srand(time(NULL))    ;
  /*iter_SUn( .2, 10);*/
  /*wilson(9.);*/
  /*
  eval_Zn(.0, 1.);
  eval_Zn(1., .0);

  zn = 3;
  eval_Zn(.0, 2.);
  eval_Zn(2., .0);
  zn = 4;
  eval_Zn(.0, 2.);
  eval_Zn(2., .0); 
  zn = 5;
  eval_Zn(.0, 3.);
  eval_Zn(3., .0);
  zn = 6;
  eval_Zn(.0, 3.);
  eval_Zn(3., .0);

  zn = 7;
  eval_Zn(.0, 4.);
  eval_Zn(4., .0);
  zn = 8;
  eval_Zn(.0, 4.);
  eval_Zn(4., .0);

  eval_U1(.0, 4.);
  eval_U1(4., .0);
  */

  eval_SUn(10., .0);
  eval_SUn(.0, 10.);
/*
  double b =5, db=.2, s;
  for (int i=0; i<30; b+=db, i++) {
    s = monte(b, ulinks, &action);
    printf("%g,  %g \n", b, s);
  } */

  return 0;
}

/*-----------------------------------------------------------------------------------------------*/

/*double db = .05;*/
void therm( double, double ); // --- for thermometer bar
int Nbeta = 10;
FILE *file; char fname[40];

void eval_Zn( double Bi, double Bf) {
  ulinks = init(1);

  double beta=Bi, S, db=(Bf-Bi)/( (double) Nbeta );
  printf("\nDETAILS: d = %d lattice (NX=%d) w/ Z_%d gauge group \n", DIM, NX, zn);

  if (Bi<Bf) {                                                      // COOLING
    sprintf(fname, "out/data/Z%d_cool_(d=%d, NX=%d).csv", zn, DIM, NX);
    printf("\n :: COOLING :: \n\n");
  }
  else if (Bi>Bf) {                                                 // HEATING
    sprintf(fname, "out/data/Z%d_heat_(d=%d, NX=%d).csv", zn, DIM, NX);
    printf("\n :: HEATING :: \n\n");
  }

  file = fopen(fname, "w+");

  fprintf(file,   "# d=%d lattice, w/ group action Z_%d \n", DIM, zn                      );
  fprintf(file,   "# MC calls %d\n", calls                                                );
  fprintf(file,   "#\n"                                                                   );
  fprintf(file,   "# beta,    action  \n"                                                 );

  for (int i=0; i<Nbeta+1; i++) 
      {   S = sweep_Zn(beta, ulinks);
          therm(S, beta ); 
          fprintf(file, "%.8f, %.8f\n", beta, S ); beta+=db;
          /*printf(       "%.8f, %.8f\n", beta, S );    }*/
      }
 
  fclose(file);                                                                             return;
}

void eval_U1( double Bi, double Bf) {
  zn=0; ulinks = init(1);

  double beta=Bi, S, db=(Bf-Bi)/( (double) Nbeta );
  printf("\nDETAILS: d = %d lattice (NX=%d) w/ U(1) gauge group \n", DIM, NX);

  if (Bi<Bf) {                                                      // COOLING
    sprintf(fname, "out/data/U(1)_cool_(d=%d, NX=%d).csv", DIM, NX);
    printf("\n :: COOLING :: \n\n");
  }
  else if (Bi>Bf) {                                                 // HEATING
    sprintf(fname, "out/data/U(1)_heat_(d=%d, NX=%d).csv", DIM, NX);
    printf("\n :: HEATING :: \n\n");
  }

  file = fopen(fname, "w+");

  fprintf(file,   "# d=%d lattice, w/ group action U(1) \n", DIM                          );
  fprintf(file,   "# MC calls %d\n", calls                                                );
  fprintf(file,   "#\n"                                                                   );
  fprintf(file,   "# beta,    action  \n"                                                 );

  for (int i=0; i<Nbeta+1; i++) 
      {   S = sweep_Zn(beta, ulinks);
          therm(S, beta ); 
          fprintf(file, "%.8f, %.8f\n", beta, S ); beta+=db;
          /*printf(       "%.8f, %.8f\n", beta, S );    }*/
      }
 
  fclose(file);                                                                             return;
}

void eval_SUn( double Bi, double Bf) {

  double beta=Bi, S, db=(Bf-Bi)/( (double) Nbeta );
  printf("\nDETAILS: d = %d lattice (NX=%d) w/ SU(%d) gauge group \n", DIM, NX, Nc);

  if (Bi<Bf) {                                                      // COOLING
    sprintf(fname, "out/data/SU(%d)_cool_(d=%d, NX=%d).csv", Nc, DIM, NX);
    ulinks = init_HOT( ); printf("\n :: COOLING :: \n\n");
  }
  else if (Bi>Bf) {                                                 // HEATING
    sprintf(fname, "out/data/SU(%d)_heat_(d=%d, NX=%d).csv", Nc, DIM, NX);
    ulinks = init_HOT( ); printf("\n :: HEATING :: \n\n");
  }

  file = fopen(fname, "w+");

  fprintf(file,   "# d=%d lattice, w/ group action SU(%d) \n",   DIM, Nc                  );
  fprintf(file,   "# MC calls %d\n", calls                                                );
  fprintf(file,   "#\n"                                                                   );
  fprintf(file,   "# beta,    action,   <L> {2,4,...}\n"                                  );

  double W1, W2, W3;
  for (int i=0; i<Nbeta+1; i++) 
      {   W1 = 0.; W2 = 0.; W3 = 0.;
          for (int trial=0; trial<10; trial++) {
            S = sweep(beta, ulinks);
            W1 += Wloop(0,1, ulinks)/10.;
            W2 += Wloop(0,2, ulinks)/10.;
            W3 += Wloop(0,3, ulinks)/10.;
          }
          fprintf(file, "%.8f, %.8f, %.8f, %.8f, %.8f\n", beta, S, W1, W2, W3 );
          therm(S, beta ); beta+=db;
          /*printf(       "%.8f, %.8f\n", beta, S );    */
      }; printf("\n");
  /*printf("<L>=%g\n", Wloop(0,2,ulinks) );*/
 
  fclose(file); free(ulinks);                                                               return;
}

void iter_SUn( double beta, int cls) {

  printf("\nDETAILS: d = %d lattice (NX=%d) w/ SU(%d) gauge group \n", DIM, NX, Nc);

  sprintf(fname, "out/data/SU(%d)_beta=%.2f_(d=%d, NX=%d).csv", Nc, beta, DIM, NX);
  ulinks = init_COLD( );

  file = fopen(fname, "w+");

  fprintf(file,   "# d=%d lattice, w/ group action SU(%d) \n",   DIM, Nc                  );
  fprintf(file,   "#\n"                                                                   );
  fprintf(file,   "# iter,    action  \n"                                                 );

  double S;
  for (int i=0; i<cls; i++) 
      {   S = sweep(beta, ulinks);
          fprintf(file, "%d, %.8f\n", i, S );
          therm(S, beta ); 
          /*printf(       "%.8f, %.8f\n", beta, S );    */
      }; printf("\n");

  printf( "W(3,4) = %g\n", Wloop(3,4,ulinks) );
 
  fclose(file); free(ulinks);                                                               return;
}

void wilson( double beta ) {

  printf("\nDETAILS: d = %d lattice (NX=%d) w/ SU(%d) gauge group \n", DIM, NX, Nc);

  sprintf(fname, "out/data/WILSON_beta=%.2f_(d=%d, NX=%d).csv", beta, DIM, NX);
  ulinks = init_HOT( );

  file = fopen(fname, "w+");

  fprintf(file,   "# d=%d lattice, w/ group action SU(%d) \n",   DIM, Nc                  );
  fprintf(file,   "#\n"                                                                   );
  fprintf(file,   "# R, T, tr(W)      \n"                                                 );

  double S;
  for (int i=0; i<20; i++) 
      {   S = sweep(beta, ulinks);
          therm(S, beta ); 
      };  printf("\n");

  double w;
  for (int R=0; R<NX; R++) for (int T=0; T<NX; T++) {
    w = Wloop( R, T, ulinks );
    printf( "W(%d,%d) = %g\n",      R, T, w );
    fprintf(file, "%d, %d, %.8f\n", R, T, w );
  }

 
  fclose(file); free(ulinks);                                                               return;
}



/*-----------------------------------------------------------------------------------------------*/

void therm( double e, double b ) {
  int width = 60;
  printf(" beta = %.1f : [", b);
  for (int i=0; i<width;++i) {
    int pos = e*width / 1.2;
         if (i<pos)  printf("-");
    else if (i==pos) printf("O");
    else             printf(" ");
  }
  printf("]  %.4f\r", e);
  fflush(stdout);
}
